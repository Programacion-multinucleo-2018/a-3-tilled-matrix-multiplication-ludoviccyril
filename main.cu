#include "hip/hip_runtime.h"
#include "main.h"

__global__ void multiply_gpu_tiling(float *a, float *b, float *c) {
  __shared__ float a_tile[TILE_DIM * TILE_DIM];
  __shared__ float b_tile[TILE_DIM * TILE_DIM];

  int n_tiles = N / TILE_DIM;

  int tile_row = blockIdx.x;
  int tile_col = blockIdx.y;

  int num_row = threadIdx.x;
  int num_col = threadIdx.y;

  float result = 0.0f;

  for (int i = 0; i < n_tiles; i++) {
    a_tile[num_row * TILE_DIM + num_col] =
        a[tile_row * N * TILE_DIM + i * TILE_DIM + N * num_row + num_col];
    b_tile[num_row * TILE_DIM + num_col] =
        b[tile_col * TILE_DIM + i * N * TILE_DIM + N * num_col + num_row];

    __syncthreads();

    for (int j = 0; j < TILE_DIM; j++) {
      result += a_tile[num_row * TILE_DIM + j] * b_tile[num_col * TILE_DIM + j];
    }

    __syncthreads();
  }

  c[tile_row * TILE_DIM * N + tile_col * TILE_DIM + num_col + num_row * N] =
      result;
}

__global__ void multiply_gpu(float *a, float *b, float *c) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  int idx = j * N + i;

  if (j < N && i < N) {
    float res = 0.0f;
    for (int k = 0; k < N; k++) {
      res += a[j * N + k] * b[k * N + i];
    }
    c[idx] = res;
  }
}

int main(int argc, char *argv[]) {

  // set up device

  hipDeviceProp_t deviceProp;
  SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0),
            "Error getting device properties");
  printf("Using device: %s\n", deviceProp.name);
  SAFE_CALL(hipSetDevice(0), "Error setting device");

  // initialize matrices on host

  float *a = (float *)calloc(N * N, sizeof(float));
  float *b = (float *)calloc(N * N, sizeof(float));
  float *c = (float *)calloc(N * N, sizeof(float));
  float *d = (float *)calloc(N * N, sizeof(float));
  float *e = (float *)calloc(N * N, sizeof(float));

  fill_matrix(a);
  fill_matrix(b);

  // assign device global memory

  float *d_a, *d_b, *d_c, *d_d;
  SAFE_CALL(hipMalloc((void **)&d_a, N * N * sizeof(float)),
            "Error allocating d_a");
  SAFE_CALL(hipMalloc((void **)&d_b, N * N * sizeof(float)),
            "Error allocating d_b");
  SAFE_CALL(hipMalloc((void **)&d_c, N * N * sizeof(float)),
            "Error allocating d_c");
  SAFE_CALL(hipMalloc((void **)&d_d, N * N * sizeof(float)),
            "Error allocating d_d");

  // transfer data from host to device

  SAFE_CALL(hipMemcpy(d_a, a, N * N * sizeof(float), hipMemcpyHostToDevice),
            "Error copying a");
  SAFE_CALL(hipMemcpy(d_b, b, N * N * sizeof(float), hipMemcpyHostToDevice),
            "Error copying b");

  // configure grid & run multiply with tiling

  dim3 block_tile(TILE_DIM, TILE_DIM);
  dim3 grid_tile(N / TILE_DIM, N / TILE_DIM);

  auto start_gpu_tile = chrono::high_resolution_clock::now();

  multiply_gpu_tiling<<<grid_tile, block_tile>>>(d_a, d_b, d_c);
  SAFE_CALL(hipDeviceSynchronize(), "Error executing kernel");

  auto end_gpu_tile = chrono::high_resolution_clock::now();

  // check for kernel errors

  SAFE_CALL(hipGetLastError(), "Error with last error");

  // configure grid & run multiply without tiling

  dim3 block(BLOCK_DIM, BLOCK_DIM);
  dim3 grid((N + block.x - 1) / block.x, (N + block.y - 1) / block.y);

  auto start_gpu = chrono::high_resolution_clock::now();

  multiply_gpu<<<grid, block>>>(d_a, d_b, d_d);
  SAFE_CALL(hipDeviceSynchronize(), "Error executing kernel");

  auto end_gpu = chrono::high_resolution_clock::now();

  // check for kernel errors

  SAFE_CALL(hipGetLastError(), "Error with last error");

  // copy results to host

  SAFE_CALL(hipMemcpy(c, d_c, N * N * sizeof(float), hipMemcpyDeviceToHost),
            "Error copying c");
  SAFE_CALL(hipMemcpy(d, d_d, N * N * sizeof(float), hipMemcpyDeviceToHost),
            "Error copying d");

  // free device global memory

  SAFE_CALL(hipFree(d_a), "Error freeing memory");
  SAFE_CALL(hipFree(d_b), "Error freeing memory");
  SAFE_CALL(hipFree(d_c), "Error freeing memory");
  SAFE_CALL(hipFree(d_d), "Error freeing memory");

  // reset device

  SAFE_CALL(hipDeviceReset(), "Error resetting");

  // multiply on host

  multiply_cpu(a, b, e);

  // check results

  cout << "CHECKING RESULTS OBTAINED WITH TILING" << endl;
  check_result(c, e);

  cout << "CHECKING RESULTS OBTAINED WITHOUT TILING" << endl;
  check_result(d, e);

  // free host memory

  free(a);
  free(b);
  free(c);
  free(d);
  free(e);

  // print results

  chrono::duration<float, std::milli> duration_gpu_tile =
      end_gpu_tile - start_gpu_tile;
  chrono::duration<float, std::milli> duration_gpu = end_gpu - start_gpu;

  cout << "WITH TILING: " << duration_gpu_tile.count() << "ms" << endl;
  cout << "WITHOUT TILING: " << duration_gpu.count() << "ms" << endl;
  cout << "SPEEDUP: " << duration_gpu.count() / duration_gpu_tile.count()
       << endl;

  return 0;
}